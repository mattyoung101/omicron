#include "hip/hip_runtime.h"
#include <opencv2/cudafeatures2d.hpp>

#include "hip/hip_runtime.h"
#include ""

#include "labeling_algorithms.h"
#include "register.h"

#include "labeling_CUDA_BUF.cuh"

// Questo algoritmo � una modifica del Block Union Find (BUF) che esegue usa la FindAndCompress al posto della 
// find usata dal BUF. La FindAndCompress aggiorna la label del pixel di partenza ad ogni iterazione della 
// procedura di ricerca della label root. Ovvert se l'albero di equivalenza � cos� costruito: 

//       A
//     /
//    B
//   /
//  C

// allora all prima iterazione aggiorno la label di C sostituendola con B e all'iterazione successiva con A.
// In questo modo se un altro thread legge il mio valore a met� trova gi� B ed evita un passaggio. Funziona meglio 
// del BUF solo a volte (raramente?). 


#define BLOCK_ROWS 16
#define BLOCK_COLS 16

using namespace cv;

namespace {

    // Only use it with unsigned numeric types
    template <typename T>
    __device__ __forceinline__ unsigned char HasBit(T bitmap, unsigned char pos) {
        return (bitmap >> pos) & 1;
    }

    //__device__ __forceinline__ void SetBit(unsigned char &bitmap, unsigned char pos) {
    //	bitmap |= (1 << pos);
    //}

    // Risale alla radice dell'albero a partire da un suo nodo n
    __device__ unsigned Find(const int *s_buf, unsigned n) {
        while (s_buf[n] != n) {
            n = s_buf[n];
        }
        return n;
    }

    __device__ unsigned FindAndCompress(int *s_buf, unsigned n) {
        unsigned id = n;
        while (s_buf[n] != n) {
            n = s_buf[n];
            s_buf[id] = n;
        }
        return n;
    }

    // Unisce gli alberi contenenti i nodi a e b, collegandone le radici
    __device__ void Union(int *s_buf, unsigned a, unsigned b) {

        bool done;

        do {

            a = Find(s_buf, a);
            b = Find(s_buf, b);

            if (a < b) {
                int old = atomicMin(s_buf + b, a);
                done = (old == b);
                b = old;
            }
            else if (b < a) {
                int old = atomicMin(s_buf + a, b);
                done = (old == a);
                a = old;
            }
            else {
                done = true;
            }

        } while (!done);

    }


    __global__ void InitLabeling(cuda::PtrStepSzi labels) {
        unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
        unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
        unsigned labels_index = row * (labels.step / labels.elem_size) + col;

        if (row < labels.rows && col < labels.cols) {
            labels[labels_index] = labels_index;
        }
    }

    __global__ void Merge(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

        unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
        unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
        unsigned img_index = row * img.step + col;
        unsigned labels_index = row * (labels.step / labels.elem_size) + col;

        if (row < labels.rows && col < labels.cols) {

            unsigned P = 0;

            char buffer[4];
            *(reinterpret_cast<int*>(buffer)) = 0;

            if (col + 1 < img.cols) {
                // This does not depend on endianness
                *(reinterpret_cast<int16_t*>(buffer)) = *(reinterpret_cast<int16_t*>(img.data + img_index));

                if (row + 1 < img.rows) {
                    *(reinterpret_cast<int16_t*>(buffer + 2)) = *(reinterpret_cast<int16_t*>(img.data + img_index + img.step));
                }
            }
            else {
                buffer[0] = img.data[img_index];

                if (row + 1 < img.rows) {
                    buffer[2] = img.data[img_index + img.step];
                }
            }

            if (buffer[0]) {
                P |= 0x777;
            }
            if (buffer[1]) {
                P |= (0x777 << 1);
            }
            if (buffer[2]) {
                P |= (0x777 << 4);
            }

            if (col == 0) {
                P &= 0xEEEE;
            }
            if (col + 1 >= img.cols) {
                P &= 0x3333;
            }
            else if (col + 2 >= img.cols) {
                P &= 0x7777;
            }

            if (row == 0) {
                P &= 0xFFF0;
            }
            if (row + 1 >= img.rows) {
                P &= 0xFF;
            }
            //else if (row + 2 >= img.rows) {                                           
            //	P &= 0xFFF;
            //}

            // P is now ready to be used to find neighbour blocks (or it should be)
            // P value avoids range errors

            if (P > 0) {

                if (HasBit(P, 0) && img.data[img_index - img.step - 1]) {
                    Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) - 2);
                }

                if ((HasBit(P, 1) && img.data[img_index - img.step]) || (HasBit(P, 2) && img.data[img_index + 1 - img.step])) {
                    Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size));
                }

                if (HasBit(P, 3) && img.data[img_index + 2 - img.step]) {
                    Union(labels.data, labels_index, labels_index - 2 * (labels.step / labels.elem_size) + 2);
                }

                if ((HasBit(P, 4) && img.data[img_index - 1]) || (HasBit(P, 8) && img.data[img_index + img.step - 1])) {
                    Union(labels.data, labels_index, labels_index - 2);
                }
            }
        }
    }

    __global__ void Compression(cuda::PtrStepSzi labels) {

        unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
        unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
        unsigned labels_index = row * (labels.step / labels.elem_size) + col;

        if (row < labels.rows && col < labels.cols) {
            FindAndCompress(labels.data, labels_index);            
        }
    }


    __global__ void FinalLabeling(const cuda::PtrStepSzb img, cuda::PtrStepSzi labels) {

        unsigned row = (blockIdx.y * BLOCK_ROWS + threadIdx.y) * 2;
        unsigned col = (blockIdx.x * BLOCK_COLS + threadIdx.x) * 2;
        unsigned labels_index = row * (labels.step / labels.elem_size) + col;
        unsigned img_index = row * (img.step / img.elem_size) + col;

        if (row < labels.rows && col < labels.cols) {

            unsigned int label = labels[labels_index] + 1;

            if (img.data[img_index])
                labels[labels_index] = label;
            else {
                labels[labels_index] = 0;
            }

            if (col + 1 < labels.cols) {
                if (img.data[img_index + 1])
                    labels[labels_index + 1] = label;
                else {
                    labels[labels_index + 1] = 0;
                }

                if (row + 1 < labels.rows) {
                    if (img.data[img_index + img.step + 1])
                        labels[labels_index + (labels.step / labels.elem_size) + 1] = label;
                    else {
                        labels[labels_index + (labels.step / labels.elem_size) + 1] = 0;
                    }
                }
            }

            if (row + 1 < labels.rows) {
                if (img.data[img_index + img.step])
                    labels[labels_index + (labels.step / labels.elem_size)] = label;
                else {
                    labels[labels_index + (labels.step / labels.elem_size)] = 0;
                }
            }

        }

    }

}

void BUF_IC::PerformLabeling() {

        d_img_labels_.create(d_img_.size(), CV_32SC1);

        grid_size_ = dim3((((d_img_.cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((d_img_.rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
        block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);

        InitLabeling <<<grid_size_, block_size_ >>> (d_img_labels_);

        //cuda::GpuMat d_expanded_connections;
        //d_expanded_connections.create(d_connections_.rows * 3, d_connections_.cols * 3, CV_8UC1);
        //ExpandConnections <<<grid_size_, block_size_ >>> (d_connections_, d_expanded_connections);
        //Mat1b expanded_connections;
        //d_expanded_connections.download(expanded_connections);
        //d_expanded_connections.release();

        //Mat1i init_labels;
        //d_block_labels_.download(init_labels);

        Merge <<<grid_size_, block_size_ >>> (d_img_, d_img_labels_);

        //Mat1i block_info_final;
        //d_img_labels_.download(block_info_final);		

        Compression <<<grid_size_, block_size_ >>> (d_img_labels_);

        FinalLabeling <<<grid_size_, block_size_ >>> (d_img_, d_img_labels_);

        // d_img_labels_.download(img_labels_);
        hipDeviceSynchronize();
    }


void BUF_IC::Alloc() {
        d_img_labels_.create(d_img_.size(), CV_32SC1);
    }

    void BUF_IC::Dealloc() {
    }

    double BUF_IC::MemoryTransferHostToDevice() {
        perf_.start();
        d_img_.upload(img_);
        perf_.stop();
        return perf_.last();
    }

    void BUF_IC::MemoryTransferDeviceToHost() {
        d_img_labels_.download(img_labels_);
    }

    void BUF_IC::AllScans() {
        grid_size_ = dim3((((d_img_.cols + 1) / 2) + BLOCK_COLS - 1) / BLOCK_COLS, (((d_img_.rows + 1) / 2) + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);
        block_size_ = dim3(BLOCK_COLS, BLOCK_ROWS, 1);


        InitLabeling <<<grid_size_, block_size_ >>> (d_img_labels_);

        //cuda::GpuMat d_expanded_connections;
        //d_expanded_connections.create(d_connections_.rows * 3, d_connections_.cols * 3, CV_8UC1);
        //ExpandConnections <<<grid_size_, block_size_ >>> (d_connections_, d_expanded_connections);
        //Mat1b expanded_connections;
        //d_expanded_connections.download(expanded_connections);
        //d_expanded_connections.release();

        //Mat1i init_labels;
        //d_block_labels_.download(init_labels);

        Merge <<<grid_size_, block_size_ >>> (d_img_, d_img_labels_);

        //Mat1i block_info_final;
        //d_img_labels_.download(block_info_final);		

        Compression <<<grid_size_, block_size_ >>> (d_img_labels_);

        FinalLabeling <<<grid_size_, block_size_ >>> (d_img_, d_img_labels_);

        hipDeviceSynchronize();
    }

    void BUF_IC::PerformLabelingWithSteps()
    {
        perf_.start();
        Alloc();
        perf_.stop();
        double alloc_timing = perf_.last();

        perf_.start();
        AllScans();
        perf_.stop();
        perf_.store(Step(StepType::ALL_SCANS), perf_.last());

        perf_.start();
        Dealloc();
        perf_.stop();
        double dealloc_timing = perf_.last();

        perf_.store(Step(StepType::ALLOC_DEALLOC), alloc_timing + dealloc_timing);
    }
