#include "hip/hip_runtime.h"
#include "in_range.cuh"

__global__ void inRange_kernel(const cv::cuda::PtrStepSz<uchar3> src, cv::cuda::PtrStepSzb dst,
                               int lbc0, int ubc0, int lbc1, int ubc1, int lbc2, int ubc2) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= src.cols || y >= src.rows) return;

    uchar3 v = src(y, x);
    if (v.x >= lbc0 && v.x <= ubc0 && v.y >= lbc1 && v.y <= ubc1 && v.z >= lbc2 && v.z <= ubc2)
        dst(y, x) = 255;
    else
        dst(y, x) = 0;
}

void inRange_gpu(cv::cuda::GpuMat &src, cv::Scalar &lowerb, cv::Scalar &upperb,
                 cv::cuda::GpuMat &dst) {
    const int m = 32;
    int numRows = src.rows, numCols = src.cols;
    if (numRows == 0 || numCols == 0) return;
    // Attention! Cols Vs. Rows are reversed
    const dim3 gridSize(ceil((float)numCols / m), ceil((float)numRows / m), 1);
    const dim3 blockSize(m, m, 1);

    inRange_kernel<<<gridSize, blockSize>>>(src, dst, lowerb[0], upperb[0], lowerb[1], upperb[1],
            lowerb[2], upperb[2]);
}